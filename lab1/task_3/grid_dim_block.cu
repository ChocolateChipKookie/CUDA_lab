#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void checkIndex(void) {
  printf("threadIdx:(%d, %d, %d) blockIdx:(%d, %d, %d) blockDim:(%d, %d, %d) "
  "gridDim:(%d, %d, %d)\n", threadIdx.x, threadIdx.y, threadIdx.z,
  blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z,
  gridDim.x,gridDim.y,gridDim.z);
}

__global__ void
vectorAdd(int *A, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        A[i] = A[i] * 2;
    }
}

int main(int argc, char **argv) {

  int nElem = 50000;
  int *h_vect = (int *)malloc(nElem * sizeof(int));
  int *d_vect = NULL;

  for (int i = 0; i < nElem; ++i)
     {
         h_vect[i] = rand();
     }

  hipMalloc((void **)&d_vect, nElem * sizeof(int));

  dim3 block(32);
  dim3 grid ((nElem+block.x-1)/block.x);
  // check grid and block dimension from host side
  printf("grid.x %d grid.y %d grid.z %d\n",grid.x, grid.y, grid.z);
  printf("block.x %d block.y %d block.z %d\n",block.x, block.y, block.z);

  hipMemcpy(d_vect, h_vect, nElem * sizeof(int), hipMemcpyHostToDevice);

  vectorAdd<<<grid, block>>>(d_vect, nElem);
  hipDeviceSynchronize();

  hipMemcpy(h_vect, d_vect, nElem * sizeof(int), hipMemcpyDeviceToHost);


  // check grid and block dimension from device side
  //checkIndex <<<grid, block>>> ();
  // reset device before you leave
  hipDeviceReset();
  free(h_vect);
  hipFree(d_vect);
  return(0);
}